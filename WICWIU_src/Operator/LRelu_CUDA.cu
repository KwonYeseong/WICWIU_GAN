
#include <hip/hip_runtime.h>
#ifdef __CUDNN__

#include "LRelu.hpp"

// template class LRelu<int>;
template class LRelu<float>;
// template class LRelu<double>;

/*!
@class LRelu cuda
*/

/*!
@brief LRelu의 ForwardPropagate 커널함수
@details ForwardPropagateOnGPU에서 호출되어 실행
@see int LRelu<DTYPE>::ForwardPropagateOnGPU(int pTime = 0)
@details 1차원으로 배열 된 block과 thread에 접근하여 연산
@param pDevInput 연산을 수행하는 input값의 GPU data
@param pDevOutput 연산의 결과인 output값을 저장할 GPU data.
@param negativeSlope input값이 0.f 이하일 때 사용하는 기울기값
@param weightDim LRelu연산의 결과값의 dimension.
*/
__global__ void ForwardPropagate_kernel(float *pDevInput, float *pDevOutput, float negativeSlope, int weightDim) {
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < weightDim; idx += blockDim.x * gridDim.x) {
          if(pDevInput[idx] > 0.f)
                pDevOutput[idx] = pDevInput[idx];
          else
                pDevOutput[idx] = negativeSlope* pDevInput[idx];
    }
}
/*!
@brief GPU에서 동작하는 ForwardPropagate 메소드.
@details GPU변수를 생성하고, 커널 함수를 실행한다.
@details noBlock는 GPU 연산시 사용되는 block의 수
@details threadsPerBlock는 한 block당 생성되는 thread개수
@details m_parameterDim는 LRelu연산의 결과값의 dimension
@details m_pDevInput, m_pDevOutput는 GPU함수 연산에 수행되는 GPU data. 각 CPU data를 GetGPUData() 호출로 GPU data생성
@see template<typename DTYPE> DTYPE *LongArray<DTYPE>::GetGPUData(unsigned int pTime)
@details ForwardPropagate_kernel 커널 함수를 호출. 커널함수이름, 블록 수, 블록당 thread 수와 GPU data를 다음과 같은 형식으로 호출.
@see __global__ void ForwardPropagate_kernel(float *pDevInput, float *pDevOutput, float negativeSlope, int weightDim)
@param pTime 연산 할 Tensor가 위치한 Time값.
@return 성공 시 TRUE.
*/
template<typename DTYPE> int LRelu<DTYPE>::ForwardPropagateOnGPU(int pTime) {
        int noBlock = 3, threadsPerBlock = 128;

        Tensor<DTYPE> *input  = this->GetInput()[0]->GetResult();
        Tensor<DTYPE> *result = this->GetResult();
        int m_parameterDim = this->GetResult()->GetCapacity();

        DTYPE *m_pDevInput  = input->GetGPUData(pTime);
        DTYPE *m_pDevOutput = result->GetGPUData(pTime);

        ForwardPropagate_kernel << < noBlock, threadsPerBlock >> > (m_pDevInput, m_pDevOutput, m_negativeSlope, m_parameterDim);

        return TRUE;
}

/*!
@brief LRelu의 BackPropagate 커널함수
@details BackPropagateOnGPU에서 호출되어 실행
@see int LRelu<DTYPE>::BackPropagateOnGPU(int pTime = 0)
@details 1차원으로 배열 된 block과 thread에 접근하여 연산
@param pDevOutput LRelu ForwardPropagate연산의 결과인 output값의 GPU data
@param pDevDelta LRelu 다음 Operator의 BackPropagate 결과 값인 delta의 GPU data.
@param pDevInputDelta 연산의 결과인 delta값을 저장할 GPU data.
@param negativeSlope output값이 0.f 이하일 때 사용하는 기울기값
@param weightDim LRelu연산의 결과값의 dimension.
*/
__global__ void BackPropagate_kernel(float *pDevOutput, float *pDevDelta, float *pDevInputDelta, float negativeSlope, int weightDim) {
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < weightDim; idx += blockDim.x * gridDim.x) {
          if(pDevOutput[idx] > 0.f)
                pDevInputDelta[idx] += pDevDelta[idx];
          else
                pDevInputDelta[idx] += negativeSlope* pDevDelta[idx];
    }
}

/*!
@brief GPU에서 동작하는 BackPropagate 메소드.
@details GPU변수를 생성하고, 커널 함수를 실행한다.
@details noBlock는 GPU 연산시 사용되는 block의 수
@details threadsPerBlock는 한 block당 생성되는 thread개수
@details m_parameterDim는 LRelu연산의 결과값의 dimension
@details m_pDevOutput, m_pDevDelta, m_pDevInputDelta는 GPU함수 연산에 수행되는 GPU data. 각 CPU data를 GetGPUData() 호출로 GPU data생성
@see template<typename DTYPE> DTYPE *LongArray<DTYPE>::GetGPUData(unsigned int pTime)
@details BackPropagate_kernel 커널 함수를 호출. 커널함수이름, 블록 수, 블록당 thread 수와 GPU data를 다음과 같은 형식으로 호출.
@see __global__ void BackPropagate_kernel(float *pDevOutput, float *pDevDelta, float *pDevInputDelta, float negativeSlope, int weightDim)
@param pTime 연산 할 Tensor가 위치한 Time값.
@return 성공 시 TRUE.
*/
template<typename DTYPE> int LRelu<DTYPE>::BackPropagateOnGPU(int pTime) {
        int noBlock = 3, threadsPerBlock = 128;

        Tensor<DTYPE> *result = this->GetResult();
        Tensor<DTYPE> *this_delta  = this->GetGradient();
        Tensor<DTYPE> *input_delta = this->GetInput()[0]->GetDelta();
        int m_parameterDim = this->GetResult()->GetCapacity();

        DTYPE *m_pDevOutput = result->GetGPUData(pTime);
        DTYPE *m_pDevDelta      = this_delta->GetGPUData(pTime);
        DTYPE *m_pDevInputDelta = input_delta->GetGPUData(pTime);

        BackPropagate_kernel << < noBlock, threadsPerBlock >> > (m_pDevOutput, m_pDevDelta, m_pDevInputDelta, m_negativeSlope, m_parameterDim);

        return TRUE;
}

#endif  // ifdef __CUDNN__
